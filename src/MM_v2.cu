#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>

using namespace std;

int save_parameter(const char* filename, int size, float *parameter);
float* get_parameter(const char* filename, int size);
__global__ void MM_v2(float*,float*,float*);

int main(){
    const char M1[] = "../data/Matrix1.bin";
    const char M2[] = "../data/Matrix2Trans.bin";
    const char Out[]="../data/MM_v2_Result.bin";

    // the parameter of the two matrix
    // int Batch = 1;
    int Batch = 128;
    int A_ROW = 128;
    int A_COL = 128;
    int B_ROW = 128;
    int B_COL = 128;

    // the number of element of matrix A, B, C 
    int nA = Batch * A_ROW * A_COL;
    int nB = Batch * B_ROW * B_COL;
    int nC = Batch * A_ROW * B_COL;

    // read data from file to the host variables
    float *h_A,*h_B,*h_C;  
    h_A = get_parameter(M1, nA);
    h_B = get_parameter(M2, nB);
    h_C = (float*)malloc(sizeof(float)*nC);

    for(int i=0;i<20;i++){
        float *d_A,*d_B,*d_C;
        hipMalloc((void **) &d_A, nA<<2);
        hipMalloc((void **) &d_B, nB<<2);
        hipMalloc((void **) &d_C, nC<<2);

        hipMemcpy(d_A, h_A, nA<<2, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, nB<<2, hipMemcpyHostToDevice);
        hipMemset((void *) d_C, 0, nC<<2);
        
        hipEvent_t start1,stop1;
        hipEventCreate(&start1);
        hipEventCreate(&stop1);
        hipEventRecord(start1,NULL);

        MM_v2<<<dim3(128,1),dim3(16,16)>>>(d_A,d_B,d_C);
        hipDeviceSynchronize();

        hipEventRecord(stop1,NULL);
        hipEventSynchronize(start1);
        hipEventSynchronize(stop1);
        
        // measure and print out the time
        float msecTotal1 = 0.0f;
        hipEventElapsedTime(&msecTotal1,start1,stop1);
        hipEventDestroy(start1);
        hipEventDestroy(stop1);
        printf("time is %lf us\n",msecTotal1*1000);

        hipMemcpy(h_C, d_C, nC<<2, hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    int cnt = save_parameter(Out,nC, h_C) * sizeof(float);
    printf("%d bytes have been writen to the file %s\n",cnt,Out);

    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}

__global__ void MM_v2(float* d_A, float* d_B, float* d_C){
    int Batch = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float output[8][8]={0};
    for(int k=0;k<64;k++){
        for(int i=0;i<128;i++){
            output[k/8][k%8] += d_A[Batch*128*128+tx*8*128 + k/8*128 +i]*d_B[Batch*128*128+ty*8*128 + k%8*128 +i];
        }
        d_C[Batch*128*128+tx*8*128 + k/8*128 + ty*8 + k%8] = output[k/8][k%8];
    }
}

int save_parameter(const char* filename, int size, float *parameter) {
    FILE* ptr = fopen(filename,"wb");

    if(!ptr){
        printf("Bad file path: %p, %s\n", ptr, strerror(errno));
        exit(0);
    }
    int cnt = fwrite(parameter,sizeof(float),size,ptr);
    fclose(ptr);
    return cnt;
}   

float* get_parameter(const char* filename, int size) {
    float* parameter = (float*)malloc(size * 4);
    if (!parameter) {
        printf("Bad Malloc\n");
        exit(0);
    }
    FILE* ptr = fopen(filename, "rb");
  
    if (!ptr) {
        printf("Bad file path: %p, %s\n", ptr, strerror(errno));
        exit(0);
    }
    fread(parameter, size * 4, 1, ptr);
  
    fclose(ptr);
    return parameter;
}
